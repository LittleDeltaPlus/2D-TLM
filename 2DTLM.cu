/***********************************************************************************************************************
 * 2D TLM for CUDA - Seámus Doran, Universiy of Nottingham 2020
 *
 * Simulates a network divided into N*N segments (nodes) of length dl
 *
 * Origin of line is matched to the source impedance i.e. no reflection from the left side of the source
 *
 * Line is excited at node Ein{ x, y } with a gaussian voltage
 *
 * Line is terminated with a short circuit to ground
 *
 * (results in an equal and opposite reflection at the end of the line)
***********************************************************************************************************************/
#include "hip/hip_runtime.h"


#include <iostream>
#include <chrono>
#include <fstream>
#include <ctime>   // for clock
#include <cmath>

#define c 299792458        // speed of light in a vacuum
#define mu0 M_PI*4e-7         // magnetic permeability in a vacuum H/m
#define eta0 c*mu0          // wave impedance in free space

using namespace std;

double** declare_array2D(int, int);                // Population function

ofstream output("output.out");       // log probe voltage at a pint on the line versus time

/**
 *  A Struct containing pointers to Data stored on Device
 */
struct dev_data{
    double* d_V1;
    double* d_V2;
    double* d_V3;
    double* d_V4;
    const double* coeff; /// A list of Coefficients, containing Z and boundary conditions
    double* out;    /// A list of voltages at the output node
    const int* d_Ein;   /// The input node
    const int* d_Eout;  /// The output node
};

/**
 * A kernel to apply a Source Voltage to a supplied input node (Ein)
 * @param dev pointers to device data
 * @param source The source to apply
 * @param N The size of Computational Domain
 */
__global__ void tlmApplySource( dev_data dev,double source, int N){
    //Apply Source
    auto tmp_idx = dev.d_Ein[0] + dev.d_Ein[1] * N;
    dev.d_V1[tmp_idx] = dev.d_V1[tmp_idx] + source;
    dev.d_V2[tmp_idx] = dev.d_V2[tmp_idx] - source;
    dev.d_V3[tmp_idx] = dev.d_V3[tmp_idx] - source;
    dev.d_V4[tmp_idx] = dev.d_V4[tmp_idx] + source;
}

/**
 *  A kernel to 'scatter' impulses based on a previously applied source
 * @param dev pointers to device data
 * @param N The size of Computational Domain
 */
__global__ void tlmScatter(dev_data dev, int N){

    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    auto idy = blockIdx.y * blockDim.y + threadIdx.y;

    auto index = idx + idy * N;

    //scatter
    double Z = dev.coeff[0];
    if ( index < N*N){
        double I = (2 * dev.d_V1[index] + 2 * dev.d_V4[index] - 2 * dev.d_V2[index] - 2 * dev.d_V3[index]) / (4 * Z);
        double V = I*Z;
        dev.d_V1[index] = dev.d_V1[index] - V;    //port1
        dev.d_V2[index] = dev.d_V2[index] + V;    //port2
        dev.d_V3[index] = dev.d_V3[index] + V;    //port3
        dev.d_V4[index] = dev.d_V4[index] - V;    //port4
    }
}

/**
 *  A kernel to propagate scattered impulses and apply boundary conditions
 * @param dev pointers to device data
 * @param N The size of Computational Domain
 */
__global__ void tlmConnect(dev_data dev, int N){
    auto idx = blockIdx.x*blockDim.x + threadIdx.x;
    auto idy = blockIdx.y*blockDim.y + threadIdx.y;

    auto index = idx + idy * N;

    //Connect
    if ( idx > 0 && index < N*N){
        auto V = dev.d_V2[index];
        dev.d_V2[index] = dev.d_V4[(idx - 1)+ idy * N];
        dev.d_V4[(idx - 1) + idy * N] = V;
    }

    if ( idy > 0 && index < N*N){
        auto V = dev.d_V1[index];
        dev.d_V1[index] = dev.d_V3[idx + (idy - 1)*N];
        dev.d_V3[idx + (idy - 1)*N] = V;
    }

    //Apply Boundaries
    // rXmin = dev.coeff[1], rXmax = dev.coeff[2]
    // rYmin = dev.coeff[3], rYmax = dev.coeff[4]

    if (idy == N-1*N && idx < N){
        dev.d_V3[idx + (N - 1)*N] = dev.coeff[4] * dev.d_V3[idx + (N - 1)*N];
        dev.d_V1[idx] = dev.coeff[3] * dev.d_V1[idx];
    }

    if (idx == N-1 && idy < N) {
        dev.d_V4[(N - 1) + idy*N] = dev.coeff[2] * dev.d_V4[(N - 1) + idy*N];
        dev.d_V2[idy*N] = dev.coeff[1] * dev.d_V2[idy*N];
    }
}

/**
 *
 * A kernel to evaluate the voltage at a supplied output node (Eout)
 * @param dev pointers to device data
 * @param n The current time-step index
 * @param N The size of Computational Domain
 */
__global__ void tlmApplyProbe(dev_data dev, int n, int N){
    auto tmp_idx = dev.d_Eout[0] + dev.d_Eout[1] * N;
    dev.out[n] = dev.d_V2[tmp_idx] + dev.d_V4[tmp_idx];
}

int main(){
    //Specify Simulation Meta Parameters
    int NX = 100;                           // dim one of nodes
    int NY = 100;                           // dim 2 of nodes
    int NT = 8192;                          // number of time steps
    double dl = 1;                          // set node line segment length in metres
    double dt = dl / (sqrt(2.) * c);     // set time step duration


    //2D mesh variables
    double** V1 = declare_array2D(NX, NY);
    double** V2 = declare_array2D(NX, NY);
    double** V3 = declare_array2D(NX, NY);
    double** V4 = declare_array2D(NX, NY);
    double v_output[NT];
    for (int n = 0; n < NT; n++){
        v_output[n] = 0;
    }

    //boundary coefficients
    double rXmin = -1;
    double rXmax = -1;
    double rYmin = -1;
    double rYmax = -1;



    // specify mesh simulation parameters
    double Z = eta0 / sqrt(2.);
    double width = 20 * dt * sqrt(2.);
    double delay = 100 * dt * sqrt(2.);
    int Ein[] = { 10,10 };
    int Eout[] = { 15,15 };

    //Group Coefficients
    double coeff[] = {Z, rXmin, rXmax, rYmin, rYmax};

    //device arrays
    double* dev_V1;
    double* dev_V2;
    double* dev_V3;
    double* dev_V4;
    double* dev_coeff;
    double* dev_output;
    int* dev_Ein;
    int* dev_Eout;


    //allocate memory on device
    auto sz = NX * NY * sizeof(double);
    hipMalloc((void**)&dev_V1, sz);
    hipMalloc((void**)&dev_V2, sz);
    hipMalloc((void**)&dev_V3, sz);
    hipMalloc((void**)&dev_V4, sz);
    hipMalloc((void**)&dev_coeff, sizeof(double)*6);
    hipMalloc((void**)&dev_output, sizeof(double)*NT);
    hipMalloc((void**)&dev_Ein, sizeof(int)*2);
    hipMalloc((void**)&dev_Eout, sizeof(int)*2);



    //copy memory areas from host to device
    hipMemcpy(dev_V1, V1, NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_V2, V2, NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_V3, V3, NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_V4, V4, NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_coeff, coeff, sizeof(double)*6, hipMemcpyHostToDevice);
    hipMemcpy(dev_Ein, Ein, sizeof(int)*2, hipMemcpyHostToDevice);
    hipMemcpy(dev_Eout, Eout, sizeof(int)*2, hipMemcpyHostToDevice);




    //Group Device Variables to simplify Kernel Calls
    dev_data dev_Data{dev_V1, dev_V2, dev_V3, dev_V4, dev_coeff, dev_output, dev_Ein, dev_Eout};
    //Determine Kernel Size
    dim3 dimBlock(10,10);
    dim3 dimGrid(ceil(NX/dimBlock.x),ceil(NY/dimBlock.y));

    //Start Timer
    auto t1 = std::chrono::high_resolution_clock::now();
    // Start of TLM algorithm
    //
    // loop over total time NT in steps of dt
    for (int n = 0; n < NT; n++)
    {
        //Calculate V Source for this delta
        double source = (1 / sqrt(2.)) * exp(-(n * dt - delay) * (n * dt - delay) / (width * width));
        //Apply the newly calculated Source
        tlmApplySource  <<<1, 1>>> (dev_Data, source, NX);
        //Apply Scatter Algorithm
        tlmScatter      <<<dimGrid, dimBlock>>> (dev_Data, NX);
        //Apply Connect Algorithm (Including Boundaries)
        tlmConnect      <<<dimGrid, dimBlock>>> (dev_Data, NX);
        //Get the Output from the mesh
        tlmApplyProbe   <<<1, 1>>> (dev_Data, n, NX);

    }
    //Get Result from Device
    hipMemcpy(v_output, dev_output, sizeof(double)*NT, hipMemcpyDeviceToHost);
    //Save output to file
    for (int n = 0; n < NT; n++){
        output << n * dt << "  " <<  v_output[n] << endl;
    }
    //End Timer
    auto t2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = t2-t1;
    // End of TLM algorithm
    //Close output file
    output.close();
    //Signify finished
    cout << "Done";
    //Calculate time / Clocks
    std::cout << "\nExecuted in:   " << (diff.count()) << "s \n";
    cin.get();


    // free memory allocated on the GPU
    hipFree(dev_V1);
    hipFree(dev_V2);
    hipFree(dev_V3);
    hipFree(dev_V4);
    hipFree(dev_output);
    hipFree(dev_coeff);
    hipFree(dev_Ein);
    hipFree(dev_Eout);

}

/**
 * A function to fill 2D arrays with 0s
 * @param NX The Array's X Dimension
 * @param NY The Array's Y Dimension
 * @return A 2D array of 0s
 */
double** declare_array2D(int NX, int NY) {
    auto** V = new double* [NX];
    for (int x = 0; x < NX; x++) {
        V[x] = new double[NY];
    }

    for (int x = 0; x < NX; x++) {
        for (int y = 0; y < NY; y++) {
            V[x][y] = 0;
        }
    }
    return V;
}