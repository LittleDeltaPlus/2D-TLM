// 1D TLM for CUDA - Department of EEE, Universiy of Nottingham 2020
//
// Simulates a line divided into NX segments (nodes) of length dl
//
// Origin of line is matched to the source impedance i.e. no reflection from the left side of the source
//
// Line is excited at node 0 with a gaussian voltage
//
// Line is terminated with a short circuit to ground 
// (results in an equal and opposite reflection at the end of the line)

#include "hip/hip_runtime.h"


#include <iostream>
#include <fstream>
#include <ctime>   // for clock
#include <cmath>

#define c 299792458        // speed of light in a vacuum
#define mu0 M_PI*4e-7         // magnetic permeability in a vacuum H/m
#define eta0 c*mu0          // wave impedance in free space

using namespace std;

double  tlmSource(double, double, double);          // excitation function
double ** declare_array2D(int, int);                // Population function

ofstream output("output.out");       // log probe voltage at a pint on the line versus time

struct dev_data{
    double* d_V1;
    double* d_V2;
    double* d_V3;
    double* d_V4;
    const double* coeff;
    double* out;
    const int* d_Ein;
    const int* d_Eout;
};
__global__ void tlmApplySource( dev_data dev,double source, int N){
    //Apply Source
    auto tmp_idx = dev.d_Ein[0] + dev.d_Ein[1] * N;
    dev.d_V1[tmp_idx] = dev.d_V1[tmp_idx] + source;
    dev.d_V2[tmp_idx] = dev.d_V2[tmp_idx] - source;
    dev.d_V3[tmp_idx] = dev.d_V3[tmp_idx] - source;
    dev.d_V4[tmp_idx] = dev.d_V4[tmp_idx] + source;
}

// TLM scatter on GPU
__global__ void tlmScatter(dev_data dev, int N, double source){

    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    auto idy = blockIdx.y * blockDim.y + threadIdx.y;

    auto index = idx + idy * N;

//    if (idx == 0 && idy == 0){
//        auto tmp_idx = dev.d_Ein[0] + dev.d_Ein[1] * N;
//        dev.d_V1[tmp_idx] = dev.d_V1[tmp_idx] + source;
//        dev.d_V2[tmp_idx] = dev.d_V2[tmp_idx] - source;
//        dev.d_V3[tmp_idx] = dev.d_V3[tmp_idx] - source;
//        dev.d_V4[tmp_idx] = dev.d_V4[tmp_idx] + source;
//    }

    //scatter
    double Z = dev.coeff[0];
   if ( index < N*N)
    {
        double I = (2 * dev.d_V1[index] + 2 * dev.d_V4[index] - 2 * dev.d_V2[index] - 2 * dev.d_V3[index]) / (4 * Z);
        double V = 2 * dev.d_V1[index] - I * Z;    //port1
        dev.d_V1[index] = V - dev.d_V1[index];
        V = 2 * dev.d_V2[index] + I * Z;         //port2
        dev.d_V2[index] = V - dev.d_V2[index];
        V = 2 * dev.d_V3[index] + I * Z;         //port3
        dev.d_V3[index] = V - dev.d_V3[index];
        V = 2 * dev.d_V4[index] - I * Z;         //port4
        dev.d_V4[index] = V - dev.d_V4[index];
   }
}

//TLM connect and apply boundary on GPU
__global__ void tlmConnect(dev_data dev, int N, int n)
{
    auto idx = blockIdx.x*blockDim.x + threadIdx.x;
    auto idy = blockIdx.y*blockDim.y + threadIdx.y;

    auto index = idx + idy * N;

    //Connect
    if ( idx > 0 && index < N*N)
    {
        auto V = dev.d_V2[index];
        dev.d_V2[index] = dev.d_V4[(idx - 1)+ idy * N];
        dev.d_V4[(idx - 1) + idy * N] = V;
    }

    if ( idy > 0 && index < N*N)
    {
        auto V = dev.d_V1[index];
        dev.d_V1[index] = dev.d_V3[idx + (idy - 1)*N];
        dev.d_V3[idx + (idy - 1)*N] = V;
    }

//    //Apply Boundaries
//    double rXmin = dev.coeff[2];
//    double rXmax = dev.coeff[3];
//    double rYmin = dev.coeff[4];
//    double rYmax = dev.coeff[5];
//
//    if (idy == N-1*N && index < N*N){
//        dev.d_V3[idx + (N - 1)*N] = rYmax * dev.d_V3[idx + (N - 1)*N];
//        dev.d_V1[idx] = rYmin * dev.d_V1[idx];
//    }
//
//    if (idx == n-1 && index < N*N) {
//        dev.d_V4[(N - 1) + idy*N] = rXmax * dev.d_V4[(N - 1) + idy*N];
//        dev.d_V2[idy*N] = rXmin * dev.d_V2[idy*N];
//    }


}

__global__ void applyBoundary(dev_data dev, int N) {
    double rXmin = dev.coeff[2];
    double rXmax = dev.coeff[3];
    double rYmin = dev.coeff[4];
    double rYmax = dev.coeff[5];

    for (int x = 0; x < N; x++) {
        dev.d_V3[x + (N - 1)*N] = rYmax * dev.d_V3[x + (N - 1)*N];
        dev.d_V1[x] = rYmin * dev.d_V1[x];
    }
    for (int y = 0; y < N; y++) {
        dev.d_V4[(N - 1) + y*N] = rXmax * dev.d_V4[(N - 1) + y*N];
        dev.d_V2[y*N] = rXmin * dev.d_V2[y*N];
    }
}

__global__ void evalutateOut(dev_data dev, int N, int n){
    auto tmp_idx = dev.d_Eout[0] + dev.d_Eout[1] * N;
    dev.out[n] = dev.d_V2[tmp_idx] + dev.d_V4[tmp_idx];
}

int main()
{


    clock_t start, end;

    int NX = 100;   // dim one of nodes
    int NY = 100;   // dim 2 of nodes
    int NT = 8192;   // number of time steps
    double dl = 1;       // set node line segment length in metres
    double dt = dl / (sqrt(2.) * c);    // set time step duration


    //2D mesh variables
    double I = 0;
    double** V1 = declare_array2D(NX, NY);
    double** V2 = declare_array2D(NX, NY);
    double** V3 = declare_array2D(NX, NY);
    double** V4 = declare_array2D(NX, NY);
    double v_output[NT];
    for (int n = 0; n < NT; n++){
        v_output[n] = 0;
    }

    double Z = eta0 / sqrt(2.);


    //boundary coefficients
    double rXmin = -1;
    double rXmax = -1;
    double rYmin = -1;
    double rYmax = -1;

    double coeff[] = {Z, I, rXmin, rXmax, rYmin, rYmax};

    //input / v_output
    double width = 20 * dt * sqrt(2.);
    double delay = 100 * dt * sqrt(2.);
    int Ein[] = { 10,10 };
    int Eout[] = { 15,15 };


    /// device arrays
    double* dev_V1;
    double* dev_V2;
    double* dev_V3;
    double* dev_V4;
    double* dev_coeff;
    double* dev_output;
    int* dev_Ein;
    int* dev_Eout;


    ///allocate memory on device
    auto sz = NX * NY * sizeof(double);
    hipMalloc((void**)&dev_V1, sz);
    hipMalloc((void**)&dev_V2, sz);
    hipMalloc((void**)&dev_V3, sz);
    hipMalloc((void**)&dev_V4, sz);
    hipMalloc((void**)&dev_coeff, sizeof(double)*6);
    hipMalloc((void**)&dev_output, sizeof(double)*NT);
    hipMalloc((void**)&dev_Ein, sizeof(int)*2);
    hipMalloc((void**)&dev_Eout, sizeof(int)*2);

    auto err = hipGetLastError();


    ///copy memory areas from host to device
    hipMemcpy(dev_V1, V1, NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_V2, V2, NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_V3, V3, NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_V4, V4, NX * NY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_coeff, coeff, sizeof(double)*6, hipMemcpyHostToDevice);
    hipMemcpy(dev_Ein, Ein, sizeof(int)*2, hipMemcpyHostToDevice);
    hipMemcpy(dev_Eout, Eout, sizeof(int)*2, hipMemcpyHostToDevice);

    err = hipGetLastError();


    // Start of TLM algorithm
    //
    // loop over total time NT in steps of dt

    dev_data dev_Data{dev_V1, dev_V2, dev_V3, dev_V4, dev_coeff, dev_output, dev_Ein, dev_Eout};

    start = clock();

    dim3 dimBlock(10,10);
    dim3 dimGrid(ceil(NX/dimBlock.x),ceil(NY/dimBlock.y));

    err = hipGetLastError();
    int i = 0;
    for (int n = 0; n < NT; n++)
    {

        double source = (1 / sqrt(2.)) * exp(-(n * dt - delay) * (n * dt - delay) / (width * width));
        tlmApplySource <<<1, 1>>> (dev_Data, source, NX);
        err = hipGetLastError();
        i = 0;
        tlmScatter <<<dimGrid, dimBlock>>> (dev_Data, NX, source);
        err = hipGetLastError();
        i = 0;
        hipDeviceSynchronize();
        tlmConnect <<<dimGrid, dimBlock>>> (dev_Data, NX, n);
        err = hipGetLastError();
        i = 0;
        applyBoundary<<<1, 1>>> (dev_Data, NX);
        err = hipGetLastError();
        i=0;
        evalutateOut<<<1, 1>>>(dev_Data, NX,n);
        err = hipGetLastError();
        i = 0;
    }
    err = hipGetLastError();
    hipMemcpy(v_output, dev_output, sizeof(double)*NT, hipMemcpyDeviceToHost);
    err = hipGetLastError();
    for (int n = 0; n < NT; n++){
        output << n * dt << "  " <<  v_output[n] << endl;
    }
    // End of TLM algorithm
    
    end = clock();

    // copy array of measured voltages from device

    // free memory allocated on the GPU
    hipFree(dev_V1);
    hipFree(dev_V2);
    hipFree(dev_V3);
    hipFree(dev_V4);
    hipFree(dev_output);
    hipFree(dev_coeff);
    hipFree(dev_Ein);
    hipFree(dev_Eout);


//    double TLM_Execution_Time = double(end - start) / double(CLOCKS_PER_SEC);
//    cout << "Time taken by TLM algorithm : " << fixed << TLM_Execution_Time << setprecision(5);
//    cout << " sec " << endl;
//    return 0;
}

double tlmSource(double time, double delay, double width)
{
    // calculate value of gaussian ecitation voltage at time point
    //E0 = (1 / sqrt(2.)) * exp(-(n * dt - delay) * (n * dt - delay) / (width * width));
    double source = exp(-1.0 * double(time - delay) * double(time - delay) / (width * width));

    // log value of gaussian voltage to file
//    gaussian_time << time << "  " << source << endl; //write source funtion to file, comment out for timing

    return source;
}

double** declare_array2D(int NX, int NY) {
    auto** V = new double* [NX];
    for (int x = 0; x < NX; x++) {
        V[x] = new double[NY];
    }

    for (int x = 0; x < NX; x++) {
        for (int y = 0; y < NY; y++) {
            V[x][y] = 0;
        }
    }
    return V;
}